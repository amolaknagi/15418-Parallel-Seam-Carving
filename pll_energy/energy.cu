#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

#define INDEX(row, col, width) (((row) * (width)) + (col))

extern float toBW(int bytes, float sec);

__global__ void
energy_kernel(uint8_t *r, uint8_t *g, uint8_t *b, double *energy, int width, int height, int blocksWide, int blocksHigh) {

    // This is index of block
    int blockX = blockIdx.x;
    int blockY = blockIdx.y;

    // This is the dimension of the block we're in
    int blockWidth = blockDim.x;
    int blockHeight = blockDim.y;

    // These are the ranges of pixels for this block
    int startX = (blockX * blockWidth);
    int endX = startX + blockWidth;
    int startY = (blockY * blockHeight);
    int endY = startY + blockHeight;

    // Get the row and col that this thread is handling.
    int col = startX + threadIdx.x;
    int row = startY + threadIdx.y;
    // printf("row:%d\n", row);

    // If we're on the border, we'll just set the energy to 1 for simplicity
    // if (startY != 16)
        // printf("here\n");
    if (row == 0 || 
        row == (height - 1) ||
        col == 0 || 
        col == (width - 1)) {
        energy[INDEX(row, col, width)] = 1;
        // printf("blockWidth: %d, blockHeight: %d, blockX: %d, blockY: %d, startX: %d, startY: %d, row: %d col: %d\n", blockWidth, blockHeight, blockX, blockY, startX, startY, row, col);
        return;
    }

    if (row > (height - 1) || col > (width - 1)) 
        return;

    uint8_t rDown = r[INDEX(row+1, col, width)];
    uint8_t gDown = g[INDEX(row+1, col, width)];
    uint8_t bDown = b[INDEX(row+1, col, width)];

    uint8_t rUp = r[INDEX(row-1, col, width)];
    uint8_t gUp = g[INDEX(row-1, col, width)];
    uint8_t bUp = b[INDEX(row-1, col, width)];

    uint8_t rLeft = r[INDEX(row, col-1, width)];
    uint8_t gLeft = g[INDEX(row, col-1, width)];
    uint8_t bLeft = b[INDEX(row, col-1, width)];

    uint8_t rRight = r[INDEX(row, col+1, width)];
    uint8_t gRight = g[INDEX(row, col+1, width)];
    uint8_t bRight = b[INDEX(row, col+1, width)];
    // printf("%d\n", rRight);

    uint8_t rdy = (rUp > rDown) ? rUp - rDown : rDown - rUp;
    uint8_t gdy = (gUp > gDown) ? gUp - gDown : gDown - gUp;
    uint8_t bdy = (bUp > bDown) ? bUp - bDown : bDown - bUp;

    uint8_t rdx = (rRight > rLeft) ? rRight - rLeft : rLeft - rRight;
    uint8_t gdx = (gRight > gLeft) ? gRight - gLeft : gLeft - gRight;
    uint8_t bdx = (bRight > bLeft) ? bRight - bLeft : bLeft - bRight;

    uint16_t rDelta = ((uint16_t)rdy) + ((uint16_t)rdx);
    uint16_t gDelta = ((uint16_t)gdy) + ((uint16_t)gdx);
    uint16_t bDelta = ((uint16_t)bdy) + ((uint16_t)bdx);

   // The maximum delta is 3 * (255 + 255)
   // which is 1530
    uint16_t delta = rDelta + gDelta + bDelta;
    double energyValue = (((double)delta) / ((double)1530));
    energy[INDEX(row, col, width)] = energyValue;


}

double *
energyCuda(uint8_t *r, uint8_t *g, uint8_t *b, int width, int height) {

    uint8_t* device_r;
    uint8_t* device_g;
    uint8_t* device_b;

    double* device_energy;

    //
    // Allocate our r g b matrices for CUDA
    //
    hipMalloc(&device_r, sizeof(uint8_t) * width * height);
    hipMalloc(&device_g, sizeof(uint8_t) * width * height);
    hipMalloc(&device_b, sizeof(uint8_t) * width * height);
    hipMalloc(&device_energy, sizeof(double) * width * height);


    // // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();



    hipMemcpy(device_r, r, sizeof(uint8_t) * width * height, hipMemcpyHostToDevice);
    hipMemcpy(device_g, g, sizeof(uint8_t) * width * height, hipMemcpyHostToDevice);
    hipMemcpy(device_b, b, sizeof(uint8_t) * width * height, hipMemcpyHostToDevice);
    hipMemset(device_energy, 0, sizeof(double) * width * height);


    // // run kernel

    int blockWidth = 16;
    int blockHeight = 16;
    dim3 blockDim(blockWidth, blockHeight);

    // Our block grid will be based on our blockWidth and blockHeight
    int blocksHigh = (int)(ceil((double)height / (double)blockHeight));
    int blocksWide = (int)(ceil((double)width / (double)blockWidth));
    dim3 gridDim(((blocksWide)), ((blocksHigh)));

    printf("Width: %d\n", width);
    printf("Height: %d\n", height);
    printf("blockWidth: %d\n", blockWidth);
    printf("blockHeight: %d\n", blockHeight);
    printf("blocksWide: %d\n", blocksWide);
    printf("blocksHigh: %d\n", blocksHigh);

    double startKernelTime = CycleTimer::currentSeconds();
    energy_kernel<<<gridDim, blockDim>>>(device_r, device_g, device_b, device_energy, width, height, blocksWide, blocksHigh);
    hipDeviceSynchronize();
    double endKernelTime = CycleTimer::currentSeconds();
    double *energy_result = (double*)malloc(sizeof(double) * width * height);
    hipMemcpy(energy_result, device_energy, sizeof(double) * width * height, hipMemcpyDeviceToHost);

    hipFree(device_r);
    hipFree(device_g);
    hipFree(device_b);
    hipFree(device_energy);

    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;
    printf("Overall: %.3f ms\n", 1000.f * overallDuration);


    return energy_result;
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
